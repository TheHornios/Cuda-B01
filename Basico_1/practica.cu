﻿/**
 * ARQUITECTURA DE COMPUTADORES
 * 2º Grado en Ingenieria Informatica
 *
 * EJEMPLO: "Dispositivos CUDA"
 * >> Propiedades de un dispositivo CUDA
 *
 * Alumno: Rodrigo Pascual Arnaiz
 * Fecha: 22/09/2022
 *
 */



/*   Includes   */
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

/*   Definicion de variables   */
#define N 16


/*   Funciones   */



/**
* Funcion: propiedadesDispositivo
* Objetivo: Mustra las propiedades del dispositvo, esta funcion
*   es ejecutada llamada y ejecutada desde el host
*
* Param: INT id_dispositivo -> ID del dispotivo
* Return: void
*/
__host__ void propiedadesDispositivo(int id_dispositivo)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, id_dispositivo);

    // calculo del numero de cores (SP)
    int cuda_cores = 0;
    int multi_processor_count = deviceProp.multiProcessorCount;
    int major = deviceProp.major;
    int minor = deviceProp.minor;


    switch (major)
    {
    case 1:
        //TESLA
        cuda_cores = 8;
        break;
    case 2:
        //FERMI
        if (minor == 0)
            cuda_cores = 32;
        else
            cuda_cores = 48;
        break;
    case 3:
        //KEPLER
        cuda_cores = 192;
        break;
    case 5:
        //MAXWELL
        cuda_cores = 128;
        break;
    case 6:
        //PASCAL
        cuda_cores = 64;
        break;
    case 7:
        //VOLTA
        cuda_cores = 64;
        break;
    case 8:
        //AMPERE
        cuda_cores = 128;
        break;
    default:
        //DESCONOCIDA
        cuda_cores = 0;
    }

    if (cuda_cores == 0 ) 
    {
        printf("!!!!!dispositivo desconocido!!!!!\n");
    }
    // presentacion de propiedades
    printf("***************************************************\n");
    printf("DISPOSIRIVO %d: %s\n", id_dispositivo, deviceProp.name);
    printf("***************************************************\n");
    printf("> Capacidad de Computo \t\t\t: %d.%d\n", major, minor);
    printf("> N. de MultiProcesadores \t\t: %d \n", multi_processor_count);
    printf("> N. de CUDA Cores (%dx%d) \t\t: %d \n", cuda_cores, multi_processor_count, cuda_cores * multi_processor_count);
    printf("> Memoria Global (total) \t\t: %zu MiB\n", deviceProp.totalGlobalMem / (1 << 20));
    printf("> Memoria Compartida (por bloque) \t: %zu KiB\n", deviceProp.sharedMemPerBlock /
        (1 << 10));
    printf("> Memoria Constante (total) \t\t: %zu KiB\n", deviceProp.totalConstMem / (1 << 10));
    printf("***************************************************\n");
}



// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
    // Obetener el dispisivo cuda
    int numero_dispositivos;
    hipGetDeviceCount(&numero_dispositivos);
    if ( numero_dispositivos != 0 ) 
    {
        printf("Se han encontrado <%d> dispositivos CUDA:\n", numero_dispositivos);
        for (int i = 0; i < numero_dispositivos; i++)
        {
            propiedadesDispositivo(i);
        }
    }
    else 
    {
        printf("!!!!!ERROR!!!!!\n");
        printf("Este ordenador no tiene dispositivo de ejecucion CUDA\n");
        printf("<pulsa [INTRO] para finalizar>");
        getchar();
        return 1;
    }




	// declaracion de arrays necesarios
	float* hst_a_matriz, * hst_b_matriz;
	float* dev_a_matriz, * dev_b_matriz;

	// reserva en el host las matriz hst
    hst_a_matriz = ( float * )malloc( N * sizeof( float ) );
    hst_b_matriz = ( float * )malloc( N * sizeof( float ) );


	// reserva en el device las matrices dev
	hipMalloc( ( void** )&dev_a_matriz, N * sizeof( float ) );
    hipMalloc( ( void** )&dev_b_matriz, N * sizeof( float ) );


	// inicializacion de datos del hst_a en el host
	srand( ( int )time( NULL ) );
	for ( int i = 0; i < N; i++ )
	{
		hst_a_matriz[ i ] = ( float )rand() / RAND_MAX;
	}

	// visualizacion de datos en el host
	printf( "ENTRADA (hst_A):\n" );
	for ( int i = 0; i < N; i++ )
	{
        if( i == N -1 ) 
        {
            printf( "%.2f\n", hst_a_matriz[ i ] );
        }
        else 
        {
            printf( "%.2f ", hst_a_matriz[ i ] );
        }
	}

    
	// copia de datos CPU a GPU
	hipMemcpy( dev_a_matriz, hst_a_matriz, N * sizeof(float), hipMemcpyHostToDevice);

    // copia de datos GPU a GPU
    hipMemcpy( dev_b_matriz, dev_a_matriz, N * sizeof(float), hipMemcpyDeviceToDevice);

    // copia de datos GPU a CPU
    hipMemcpy( hst_b_matriz, dev_b_matriz, N * sizeof(float), hipMemcpyDeviceToHost);

    // visualizacion de datos en el Device
    printf( "SALIDA (hst_b):\n" );
    for ( int i = 0; i < N; i++ )
    {
        if ( i == N - 1 )
        {
            printf( "%.2f\n", hst_b_matriz[i] );
        }
        else
        {
            printf( "%.2f ", hst_b_matriz[i] );
        }
    }

     // salida
	time_t fecha;
	time( &fecha );
	printf( "***************************************************\n" );
	printf( "Programa ejecutado el: %s\n", ctime( &fecha ) );
	printf( "<pulsa [INTRO] para finalizar>" );
	getchar();
	return 0;
}

